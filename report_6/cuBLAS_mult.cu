#include "hip/hip_runtime.h"
// cuBLAS matrix multiplication - lab 6 
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdlib.h>
#include <time.h>
#include <stdio.h>
#include <math.h>
#include <sys/time.h>

// Allocates a matrix with random float entries.
void randomInit(float *data, int size)
{
    for (int i = 0; i < size; ++i)
        data[i] = rand() / (float)RAND_MAX;
}

double cpuTimer()
{
	struct timeval clock;
	gettimeofday(&clock, NULL);
	return ((double)clock.tv_sec + (double)clock.tv_usec * 1e-6);
}

int main() {

	int rows_A, cols_A, rows_B, cols_B;
	rows_A = cols_A = rows_B = cols_B = 1 << 10;

	// (rows_A x cols_A) * (rows_B x cols_B) = (rows_A x cols_B) && cols_A == rows_B
	if( cols_A != rows_B )
    {
       printf("ERROR: Matrix sizes do not match!\n");
       exit(-1);
    }

	int rows_C = rows_A;
	int cols_C = cols_B;


	// Problem size
	int size_A = rows_A * cols_A;
	int size_B = rows_B * cols_B;
	int size_C = rows_C * cols_C;
	size_t bytes_A = size_A * sizeof(float);
	size_t bytes_B = size_B * sizeof(float);
	size_t bytes_C = size_C * sizeof(float);

	// Pointers to host and device
	float *h_a, *h_b, *h_c;
	float *d_a, *d_b, *d_c;

	// Allocate memory for hosts
	h_a = (float*)malloc(bytes_A);
	h_b = (float*)malloc(bytes_B);
	h_c = (float*)malloc(bytes_C);

	// Alocate memory for devices
	hipMalloc(&d_a, bytes_A);
	hipMalloc(&d_b, bytes_B);
	hipMalloc(&d_c, bytes_C);

	// set seed for rand()
    srand(2006);

    // initialize host memory
    randomInit(h_a, size_A);
    randomInit(h_b, size_B);

	// hipMemcpy from host to device
	hipMemcpy(h_a, d_a, bytes_A, hipMemcpyHostToDevice);
	hipMemcpy(h_b, d_b, bytes_B, hipMemcpyHostToDevice);
	hipMemcpy(h_c, d_c, bytes_C, hipMemcpyHostToDevice);

	// cuBLAS handle
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	// Scalaing factors
	float alpha = 1.0f;
	float beta = 0.0f;

	double ti = cpuTimer();
	// c = (alpha*a) * b + (beta*c)
	// (m X n) * (n X k) = (m X k)    || (rows_A x cols_A) * (rows_B x cols_B) = (rows_A x cols_B)
	// hipblasSgemm(handle, operation, operation, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc)
	// lda = rows_A = ldc 		ldb = cols_B
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, rows_A, cols_A, cols_B, &alpha, d_a, rows_A, d_b, cols_B, &beta, d_c, rows_A);
	double elapsed = cpuTimer - ti;

	printf("The time of matrix multiplication using cuBLAS library is equal to: %.6f", elapsed);
	
	// Destroying the handle
	hipblasDestroy(handle)
	
	// Copying back the three matrices is optionall. For example if we want to print the result.
	hipMemcpy(h_a, d_a, bytes_A, hipMemcpyDeviceToHost);
	hipMemcpy(h_b, d_b, bytes_B, hipMemcpyDeviceToHost);
	hipMemcpy(h_c, d_c, bytes_C, hipMemcpyDeviceToHost);


	//Free GPU memory
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	// Free CPU memory
	free(h_a);
	free(h_b);
	free(h_c);


	return 0;
}

