#include "hip/hip_runtime.h"
/*
 * matrix_mult.cu
 *
 *  Created on: Nov 14, 2019
 *      Author: cuda-s18
 */

#include <stdio.h>

#include <assert.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>


void initWith(float *a, int N)
{
  for(int i = 0; i < N; ++i)
  {
          for( int j = 0; j < N; ++j){
                a[i*N+j] = (i+j);
          }
  }
}
__global__ void MatrixMulKernel(float* M, float* N, float* P, int Width) {

        // Calculate the row index of the P element and M
                int Row = blockIdx.y*blockDim.y+threadIdx.y;

        // Calculate the column index of P and N
                int Col = blockIdx.x*blockDim.x+threadIdx.x;
                        if ((Row < Width) && (Col < Width)) {
                                float Pvalue = 0;

        // each thread computes one element of the block sub-matrix
                                for (int k = 0; k < Width; ++k) {
                                        Pvalue += M[Row*Width+k]*N[k*Width+Col];
                                }
                        P[Row*Width+Col] = Pvalue;
                        }
        }

int main()
{
  const int N = 2<<8;
  size_t size = N * N * sizeof(float);

  float *a;
  float *b;
  float *c;

  hipMallocManaged(&a, size);
  hipMallocManaged(&b, size);
  hipMallocManaged(&c, size);

  int deviceId;
  hipGetDevice(&deviceId);

  hipMemPrefetchAsync(a, size, deviceId);
  hipMemPrefetchAsync(a, size, hipCpuDeviceId);
  hipMemPrefetchAsync(b, size, deviceId);
  hipMemPrefetchAsync(b, size, hipCpuDeviceId);
  hipMemPrefetchAsync(c, size, deviceId);
  hipMemPrefetchAsync(c, size, hipCpuDeviceId);

  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, deviceId);

  int multiProcessorCount = props.multiProcessorCount;

  size_t threadsPerBlock;
  size_t numberOfBlocks;

  threadsPerBlock = 1024;
  numberOfBlocks = multiProcessorCount/10;

  initWith(a, N);
  initWith(b, N);

  //stuff to register elapsed time
  float el_time;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start, 0);

  hipError_t addVectorsErr;
  hipError_t asyncErr;

  MatrixMulKernel<<<1, 1>>>(a,b, c, N);

  hipEventRecord(stop,0);
  hipEventSynchronize(stop);


  hipEventElapsedTime(&el_time, start, stop);
  printf("Time elapsed on single-threaded matrix multiplication: %f", el_time);
 
 //multi
  float el_time = 0.0f;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start, 0);

  hipError_t addVectorsErr;
  hipError_t asyncErr;

  MatrixMulKernel<<<numberOfBlocks, threadsPerBlock>>>(a,b, c, N);

  hipEventRecord(stop,0);
  hipEventSynchronize(stop);


  hipEventElapsedTime(&el_time, start, stop);
  printf("Time elapsed on multi-threaded matrix multiplication: %f", el_time);
  
  
  addVectorsErr = hipGetLastError();
  if(addVectorsErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(addVectorsErr));

  asyncErr = hipDeviceSynchronize();
  if(asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));

  hipFree(a);
  hipFree(b);
  hipFree(c);
}


